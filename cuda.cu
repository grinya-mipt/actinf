#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <thrust/iterator/counting_iterator.h>
#include "vector.h"
#include "matrix.h"


#include <hip/hip_runtime.h>
#include <hipblas.h>

typedef util::Vector<3, float> Vec3f;

#define MPREC float
//#define MPREC double
typedef thrust::host_vector<float> HVecF;
typedef thrust::host_vector<unsigned int> HVecU;
typedef thrust::host_vector<unsigned char> HVecB;
typedef thrust::device_vector<float> DVecF;
typedef thrust::device_vector<unsigned int> DVecU;
typedef thrust::device_vector<unsigned char> DVecB;

using std::cout;
using std::cerr;
using std::endl;

// Move Point based on forces
struct movePointsOP
{
  Vec3f *pos, *vel, *force, *box;
  MPREC damp, friction, threshold, dt;

  movePointsOP(Vec3f *_pos, Vec3f *_vel, Vec3f *_force, Vec3f *_box, MPREC _damp, MPREC _friction, MPREC _threshold, MPREC _dt) : 
    pos(_pos), vel(_vel), force(_force), box(_box), damp(_damp), friction(_friction), threshold(_threshold), dt(_dt) {}

  __device__
    void operator()(const int vtx) const 
    { 
      vel[vtx] += force[vtx] / damp;
      pos[vtx] += dt * vel[vtx];

      for(int i = 0; i < 3; i ++) {
        if(pos[vtx][i] > box[0][i]) {
          pos[vtx] -= vel[vtx] * dt;
          pos[vtx][i] = box[0][i];
          vel[vtx][i] = 0;
          pos[vtx] += dt * vel[vtx];
        }
        if(pos[vtx][i] < -box[0][i]) {
          pos[vtx] -= dt * vel[vtx];
          pos[vtx][i] = -box[0][i];
          vel[vtx][i] = 0;
          pos[vtx] += dt * vel[vtx];
        }
      }
      return;
    }
};

// Find forces
struct findForcesOP
{
  Vec3f *pos, *force;
  float *restlen, kr, dt;
  unsigned int *nbhd, nbs;


  findForcesOP(unsigned int *_nbhd, Vec3f *_pos, Vec3f *_force, float *_restlen, Vec3f *_gravity, float _kr, float _dt, unsigned int _nbs) : 
    nbhd(_nbhd), pos(_pos), force(_force), restlen(_restlen), kr(_kr), dt(_dt), nbs(_nbs) {}

  __device__
    void operator()(const int vtx) const 
    { 
      unsigned int edg = vtx * nbs;
      Vec3f dir;

      for(unsigned int i = 0; i < nbs; i++, edg++)
        if(nbhd[edg] != vtx) {
          dir = pos[nbhd[edg]] - pos[vtx];
          float len = norm(dir);
          dir /= len;
          force[vtx] += dir * (len - restlen[edg]) * kr;
        }

      return;
    }
};

// Error checking
hipError_t checkCudaError(char *msg)
{
  hipError_t cuerr = hipGetLastError();
  if(cuerr != hipSuccess)
    fprintf(stderr, "CUDA Error %d: %s\n", cuerr, msg);
  return(cuerr);
}

static const unsigned int maxn = 100;

// Distributed objects
static DVecU* Nhbd[maxn];  // Neighborhood information
struct NInfo {
  unsigned int n;    // Number of vertices in graph 
  unsigned int nbs;    // Max number of neighbors
} NInfo[maxn];
static DVecB* VData[maxn];  // Vertex data
static DVecB* EData[maxn];  // Edge data

extern "C" {

  int initDNC()
  {
    static bool initialized = false;
    if(!initialized) {
      for(int i = 0; i < maxn; i++) {
        VData[i] = 0; EData[i] = 0; Nhbd[i] = 0; 
        NInfo[i].n = NInfo[i].nbs = 0;
      }
      initialized = true;

      // Initialize cuda
      int count, i;
      hipGetDeviceCount(&count);
      if(count == 0) {
        cerr << "There is no cuda device." << endl;
        return(1);
      }

      for(i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
          if(prop.major >= 1) {
            break;
          }
        }
      }
      if(i == count) {
        cerr << "There is no device supporting CUDA." << endl;
        return(2);
      }
      hipSetDevice(i);
      checkCudaError("Setting cuda device");
    }
    return(0);
  }

  int allocDeviceDNC(unsigned int idx, unsigned int n, unsigned int nbs)
  {
    if(idx > maxn)
      return(1);

    if(!Nhbd[idx] || NInfo[idx].n != n || NInfo[idx].nbs != nbs) {
      if(Nhbd[idx])
        delete(Nhbd[idx]);
      Nhbd[idx] = 0;
      if(n * nbs > 0)
        Nhbd[idx] = new DVecU(n * nbs);
      NInfo[idx].n = n;
      NInfo[idx].nbs = nbs;
    }
    if(!Nhbd[idx]) {
      cout << "allocDeviceDNC::Error allocating buffer" << endl;
      return(2);
    }

    return(0);
  }

  int allocDeviceDVC(unsigned int nidx, unsigned int idx, unsigned int sz)
  {
    if(idx > maxn)
      return(1);

    if(!VData[idx] || (VData[idx])->size() != NInfo[nidx].n * sz) {
      if(VData[idx])
        delete(VData[idx]);
      VData[idx] = 0;
      if(NInfo[nidx].n * sz > 0)
        VData[idx] = new DVecB(NInfo[nidx].n * sz);
    }
    if(!VData[idx]) {
      cout << "allocDeviceDVC::Error allocating buffer" << endl;
      return(2);
    }

    return(0);
  }

  int allocDeviceDEC(unsigned int nidx, unsigned int idx, unsigned int sz)
  {
    if(idx > maxn)
      return(1);

    if(!EData[idx] || (EData[idx])->size() != NInfo[nidx].n * NInfo[nidx].nbs * sz) {
      if(EData[idx])
        delete(EData[idx]);
      EData[idx] = 0;
      if(NInfo[nidx].n * NInfo[nidx].nbs * sz > 0)
        EData[idx] = new DVecB(NInfo[nidx].n * NInfo[nidx].nbs * sz);
    }
    if(!EData[idx]) {
      cout << "allocDeviceDNC::Error allocating buffer" << endl;
      return(2);
    }

    return(0);
  }

  int copyToDeviceDNC(unsigned int r, HVecU &nhbd)
  {
    if(r > maxn || !Nhbd[r] || nhbd.size() != (Nhbd[r])->size()) {
      cout << "copyToDeviceDNC::Error r:" << r << " size:" << nhbd.size() << " alloc size:" << (Nhbd[r])->size() << endl;
      return(1);
    }

    *Nhbd[r] = nhbd;
    return(0);
  }

  int copyToDeviceDVC(unsigned int r, HVecB &vdata)
  {
    if(r > maxn || !VData[r] || vdata.size() != (VData[r])->size()) {
      cout << "copyToDeviceDVC::Error r:" << r << " size:" << vdata.size() << " alloc size:" << (VData[r])->size() << endl;
      return(1);
    }
    *VData[r] = vdata;
    return(0);
  }

  int copyToHostDVC(unsigned int r, HVecB &vdata)
  {
    if(r > maxn || !VData[r] || vdata.size() != (VData[r])->size()) {
      cout << "copyToHostDVC::Error r:" << r << " size:" << vdata.size() << " alloc size:" << (VData[r])->size() << endl;
      return(1);
    }
    vdata = *VData[r];
    return(0);
  }

  int copyToDeviceDEC(unsigned int r, HVecB &edata)
  {
    if(r > maxn || !EData[r] || edata.size() != (EData[r])->size()) {
      cout << "copyToDeviceDEC::Error r:" << r << " size:" << edata.size() << " alloc size:" << (EData[r])->size() << endl;
      return(1);
    }
    *EData[r] = edata;
    return(0);
  }

  int massSpring(unsigned int nb, unsigned int pos, unsigned int vel, unsigned int force, unsigned int restlen, 
      float *box, float kr, float damp, float friction, float threshold, float dt, unsigned int steps)
  {
    if(nb > maxn || !Nhbd[nb] || pos > maxn || !VData[pos] || vel > maxn || !VData[vel] || force > maxn || !VData[force]) 
      return(1);

    DVecF TBox(box, box + 3);
    Vec3f *Box = (Vec3f *)(&TBox[0]).get();
    Vec3f *Pos = (Vec3f *)(&(*VData[pos])[0]).get();
    Vec3f *Vel = (Vec3f *)(&(*VData[vel])[0]).get();
    Vec3f *Force = (Vec3f *)(&(*VData[force])[0]).get();
    MPREC *Restlen = (MPREC *)(&(*EData[restlen])[0]).get();
    unsigned int *Nb = (unsigned int *)(&(*Nhbd[nb])[0]).get();
/*
    thrust::counting_iterator<int, thrust::device_space_tag> first(0);
    thrust::counting_iterator<int, thrust::device_space_tag> last(NInfo[nb].n);

    findForcesOP ffOP(Nb, Pos, Force, Restlen, Gravity, kr, dt, NInfo[nb].nbs);
    movePointsOP mpOP(Pos, Vel, Force, Box, damp, friction, threshold, dt);

    for(unsigned int i = 0; i < steps; i++) {
      thrust::for_each(first, last, ffOP);
      hipDeviceSynchronize();
      thrust::for_each(first, last, mpOP);
      hipDeviceSynchronize();
    }*/
    return(0);
  }
}
